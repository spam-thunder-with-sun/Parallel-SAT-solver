using namespace std;

//Librerie standard
#include <vector>
#include <iostream>
#include <string>
#include <cmath>
#include <bitset>
#include <stdio.h>
//Librerie per il calcolo parallelo
#include <hip/hip_runtime.h>
#include "hipblas.h"
//Librerie fatte
#include "read_data.h"
#include "constant.h"

#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}

int cublas_test()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (float)(i * N + j + 1);
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);

    return EXIT_SUCCESS;
}

int main(int argc, char *argv[]) 
{
    string filename = "../input/dimacs/small.cnf";

    if(argc > 1)
        filename = argv[1];

    INT_TYPE literals, clauses;
    vector<vector<INT_TYPE>> matrix;
    std::tie(literals, clauses, matrix) = readDimacsFile2Vec(filename);
    print_matrix(literals, clauses, matrix);

    cublas_test();

    return 0;
}
