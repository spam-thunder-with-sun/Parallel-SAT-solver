using namespace std;

//Librerie standard
#include <vector>
#include <iostream>
#include <string>
#include <cmath>
#include <bitset>
#include <stdio.h>
//Librerie per il calcolo parallelo
#include <hip/hip_runtime.h>
#include "hipblas.h"
//Librerie fatte
#include "read_data.h"
#include "constant.h"

#define M 6
#define N 5

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}

int cublas_test()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (float)(i * N + j + 1);
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);

    return EXIT_SUCCESS;
}

int cublas_allocation(INT_TYPE literals, INT_TYPE clauses, DATA_TYPE* matrix)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    DATA_TYPE* devPtr;

    if (!matrix) {
        cerr << "Host memory allocation failed" << endl;
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&devPtr, literals*clauses*sizeof(*matrix)<<1);
    if (cudaStat != hipSuccess) {
        cerr << "Device memory allocation failed: " << cudaStat << endl;
        return EXIT_FAILURE;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        cerr << "CUBLAS initialization failed: " << stat << endl;
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix (literals<<1, clauses, sizeof(*matrix), matrix, literals<<1, devPtr, literals<<1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        cerr << "Data download failed: " << stat << endl;
        hipFree (devPtr);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    /*
    modify (handle, devPtr, literals<<1, N, 1, 2, 16.0f, 12.0f);
    
    */

    stat = hipblasGetMatrix (literals<<1, clauses, sizeof(*matrix), devPtr, literals<<1, matrix, literals<<1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        cerr << "Data upload failed: " << stat << endl;
        hipFree (devPtr);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipFree(devPtr);
    hipblasDestroy(handle);

    return EXIT_SUCCESS;
}

int main(int argc, char *argv[]) 
{
    string filename = "../input/dimacs/small.cnf";

    if(argc > 1)
        filename = argv[1];

    INT_TYPE literals, clauses;
    DATA_TYPE* matrix;
    std::tie(literals, clauses, matrix) = readDimacsFile2Column(filename);
    print_matrix(literals, clauses, matrix);

    if(cublas_allocation(literals, clauses, matrix))
        cout << "Cublas test failed" << endl;
    else
        cout << "Cublas test passed" << endl;

    cout << endl;
    cout << "After cublas" << endl;
    print_matrix(literals, clauses, matrix);

    free(matrix);

    return 0;
}
